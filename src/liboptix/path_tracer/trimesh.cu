#include "hip/hip_runtime.h"

/*
* Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
*
* NVIDIA Corporation and its licensors retain all intellectual property and proprietary
* rights in and to this software, related documentation and any modifications thereto.
* Any use, reproduction, disclosure or distribution of this software and related
* documentation without an express license agreement from NVIDIA Corporation is strictly
* prohibited.
*
* TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
* AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
* INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
* PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
* SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
* LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
* BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
* INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
* SUCH DAMAGES
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
//#include <optix_world.h>


using namespace optix;

// This is to be plugged into an RTgeometryInstance object to represent
// a triangle mesh from mitsuba with a vertex info buffer containing
// vertex position, normal (optional), texture coordinates (optional), 
// UV Tangents (optional) and color (optional).

rtBuffer<float> vertex_info_buffer;
rtBuffer<uint3> index_buffer;

rtDeclareVariable(int, stride, , );
rtDeclareVariable(int, normal_offset, , );
rtDeclareVariable(int, texCoord_offset, , );
rtDeclareVariable(int, UVTangent_offset, , );
rtDeclareVariable(int, color_offset, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void mesh_intersect(int primIdx)
{	
	uint3 v_idx = index_buffer[primIdx];

	float3 p0 = make_float3(vertex_info_buffer[v_idx.x*stride], vertex_info_buffer[v_idx.x*stride+1], vertex_info_buffer[v_idx.x*stride+2]);
	float3 p1 = make_float3(vertex_info_buffer[v_idx.y*stride], vertex_info_buffer[v_idx.y*stride+1], vertex_info_buffer[v_idx.y*stride+2]);
	float3 p2 = make_float3(vertex_info_buffer[v_idx.z*stride], vertex_info_buffer[v_idx.z*stride+1], vertex_info_buffer[v_idx.z*stride+2]);

	// Intersect ray with triangle
	float3 n;
	float  t, beta, gamma;
	if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {

		if (rtPotentialIntersection(t)) {


			if (normal_offset==0) {
				shading_normal = normalize(n);
				//shading_normal = make_float3(1.0,0.0,0.0);
			} else {
				float3 n0 = make_float3(vertex_info_buffer[v_idx.x*stride + normal_offset],
										vertex_info_buffer[v_idx.x*stride + normal_offset + 1],
										vertex_info_buffer[v_idx.x*stride + normal_offset + 2]);
				float3 n1 = make_float3(vertex_info_buffer[v_idx.y*stride + normal_offset],
										vertex_info_buffer[v_idx.y*stride + normal_offset + 1],
										vertex_info_buffer[v_idx.y*stride + normal_offset + 2]);
				float3 n2 = make_float3(vertex_info_buffer[v_idx.z*stride + normal_offset],
										vertex_info_buffer[v_idx.z*stride + normal_offset + 1],
										vertex_info_buffer[v_idx.z*stride + normal_offset + 2]);
				shading_normal = normalize(n1*beta + n2*gamma + n0*(1.0f - beta - gamma));
				//shading_normal = make_float3(0.0, 1.0, 0.0);
			}
			geometric_normal = normalize(n);

			//int3 t_idx = tindex_buffer[primIdx];
			//if (texcoord_buffer.size() == 0 || t_idx.x < 0 || t_idx.y < 0 || t_idx.z < 0) {
			//	texcoord = make_float3(0.0f, 0.0f, 0.0f);
			//} else {
			//	float2 t0 = texcoord_buffer[t_idx.x];
			//	float2 t1 = texcoord_buffer[t_idx.y];
			//	float2 t2 = texcoord_buffer[t_idx.z];
			//	texcoord = make_float3(t1*beta + t2*gamma + t0*(1.0f - beta - gamma));
			//}

			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void mesh_bounds(int primIdx, float result[6])
{
	const uint3 v_idx = index_buffer[primIdx];

	const float3 v0 = make_float3(vertex_info_buffer[v_idx.x*stride], vertex_info_buffer[v_idx.x*stride + 1], vertex_info_buffer[v_idx.x*stride + 2]);
	const float3 v1 = make_float3(vertex_info_buffer[v_idx.y*stride], vertex_info_buffer[v_idx.y*stride + 1], vertex_info_buffer[v_idx.y*stride + 2]);
	const float3 v2 = make_float3(vertex_info_buffer[v_idx.z*stride], vertex_info_buffer[v_idx.z*stride + 1], vertex_info_buffer[v_idx.z*stride + 2]);

	const float  area = length(cross(v1 - v0, v2 - v0));

	optix::Aabb* aabb = (optix::Aabb*)result;

	if (area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf(fminf(v0, v1), v2);
		aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
	} else {
		aabb->invalidate();
	}
}

